
#include <hip/hip_runtime.h>
/* -*- mode: c++ -*- */

#define UNROLL9(F)				\
	F(0);					\
	F(1);					\
	F(2);					\
	F(3);					\
	F(4);					\
	F(5);					\
	F(6);					\
	F(7);					\
	F(8);					\


#define UNROLL8x3x3(F)				\
	F(0,0,0);				\
	F(0,0,1);				\
	F(0,0,2);				\
	F(0,1,0);				\
	F(0,1,1);				\
	F(0,1,2);				\
	F(0,2,0);				\
	F(0,2,1);				\
	F(0,2,2);				\
						\
	F(1,0,0);				\
	F(1,0,1);				\
	F(1,0,2);				\
	F(1,1,0);				\
	F(1,1,1);				\
	F(1,1,2);				\
	F(1,2,0);				\
	F(1,2,1);				\
	F(1,2,2);				\
						\
	F(2,0,0);				\
	F(2,0,1);				\
	F(2,0,2);				\
	F(2,1,0);				\
	F(2,1,1);				\
	F(2,1,2);				\
	F(2,2,0);				\
	F(2,2,1);				\
	F(2,2,2);				\
						\
	F(3,0,0);				\
	F(3,0,1);				\
	F(3,0,2);				\
	F(3,1,0);				\
	F(3,1,1);				\
	F(3,1,2);				\
	F(3,2,0);				\
	F(3,2,1);				\
	F(3,2,2);				\
						\
	F(4,0,0);				\
	F(4,0,1);				\
	F(4,0,2);				\
	F(4,1,0);				\
	F(4,1,1);				\
	F(4,1,2);				\
	F(4,2,0);				\
	F(4,2,1);				\
	F(4,2,2);				\
						\
	F(5,0,0);				\
	F(5,0,1);				\
	F(5,0,2);				\
	F(5,1,0);				\
	F(5,1,1);				\
	F(5,1,2);				\
	F(5,2,0);				\
	F(5,2,1);				\
	F(5,2,2);				\
						\
	F(6,0,0);				\
	F(6,0,1);				\
	F(6,0,2);				\
	F(6,1,0);				\
	F(6,1,1);				\
	F(6,1,2);				\
	F(6,2,0);				\
	F(6,2,1);				\
	F(6,2,2);				\
						\
	F(7,0,0);				\
	F(7,0,1);				\
	F(7,0,2);				\
	F(7,1,0);				\
	F(7,1,1);				\
	F(7,1,2);				\
	F(7,2,0);				\
	F(7,2,1);				\
	F(7,2,2);				\

#define UNROLL8(F)				\
	F(0);					\
	F(1);					\
	F(2);					\
	F(3);					\
	F(4);					\
	F(5);					\
	F(6);					\
	F(7);					\


#define UNROLL8x3(F)				\
	F(0,0);					\
	F(0,1);					\
	F(0,2);					\
	F(0,3);					\
	F(0,4);					\
	F(0,5);					\
	F(0,6);					\
	F(0,7);					\
						\
	F(1,0);					\
	F(1,1);					\
	F(1,2);					\
	F(1,3);					\
	F(1,4);					\
	F(1,5);					\
	F(1,6);					\
	F(1,7);					\
						\
	F(2,0);					\
	F(2,1);					\
	F(2,2);					\
	F(2,3);					\
	F(2,4);					\
	F(2,5);					\
	F(2,6);					\
	F(2,7);					\


#define UNROLL10x3(F)				\
	F(0,0);					\
	F(0,1);					\
	F(0,2);					\
	F(0,3);					\
	F(0,4);					\
	F(0,5);					\
	F(0,6);					\
	F(0,7);					\
	F(0,8);					\
	F(0,9);					\
						\
	F(1,0);					\
	F(1,1);					\
	F(1,2);					\
	F(1,3);					\
	F(1,4);					\
	F(1,5);					\
	F(1,6);					\
	F(1,7);					\
	F(1,8);					\
	F(1,9);					\
						\
	F(2,0);					\
	F(2,1);					\
	F(2,2);					\
	F(2,3);					\
	F(2,4);					\
	F(2,5);					\
	F(2,6);					\
	F(2,7);					\
	F(2,8);					\
	F(2,9);					\


#define BLOCK_SIZE 8

extern "C" __global__ void
filter(const float * __restrict__ packed_input,
       int nInputPlanes,
       float * __restrict__ packed_output,
       int nOutputPlanes,
       const float * __restrict__ biases,
       unsigned int hsz,
       unsigned int wsz,
       const float * __restrict__ weight)
{
	extern __shared__ float shared_buf[];

	unsigned int yi = blockIdx.x;

	size_t in_step = wsz * nInputPlanes;
	const float *inp = packed_input;
	inp += yi * in_step;

	const float *in0p = inp - in_step;
	if (yi == 0) {
		in0p = inp;
	}
	const float *in1p = inp;

	const float *in2p = inp + in_step;
	if (yi == hsz-1) {
		in2p = in1p;
	}

	const float *in01 = in0p;
	const float *in11 = in1p;
	const float *in21 = in2p;

	float *shared_ptr = shared_buf;
	float *in_block0_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);
	float *in_block1_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);
	float *in_block2_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);

	float *in_block0 = in_block0_base + nInputPlanes;
	float *in_block1 = in_block1_base + nInputPlanes;
	float *in_block2 = in_block2_base + nInputPlanes;
	int lid = threadIdx.x;
	float bv0 = biases[lid*2+0];
	float bv1 = biases[lid*2+1];

	for (int xi0=0; xi0<wsz; xi0+=BLOCK_SIZE) {
		/*for (unsigned int op=0; op<nOutputPlanes; op++) thread */
		{
			int op = lid*2;
			int rem = wsz - xi0;
			__syncthreads();
			if (lid < nInputPlanes/2) {
				int bi;
				int lid2 = lid*2;
				for (bi=0; bi<BLOCK_SIZE; bi++) {
					int xi = xi0 + bi;
					if (xi == wsz) {
						break;
					}

					/* load to shared */
					*(float2*)&in_block0[bi*nInputPlanes + lid2] = *(float2*)&in01[xi*nInputPlanes + lid2];
					*(float2*)&in_block1[bi*nInputPlanes + lid2] = *(float2*)&in11[xi*nInputPlanes + lid2];
					*(float2*)&in_block2[bi*nInputPlanes + lid2] = *(float2*)&in21[xi*nInputPlanes + lid2];
				}

				{
					int xi = xi0 + bi;
					if (xi == wsz) {
						*(float2*)&in_block0[bi*(int)nInputPlanes + lid2] = *(float2*)&in01[(xi-1)*(int)nInputPlanes + lid2];
						*(float2*)&in_block1[bi*(int)nInputPlanes + lid2] = *(float2*)&in11[(xi-1)*(int)nInputPlanes + lid2];
						*(float2*)&in_block2[bi*(int)nInputPlanes + lid2] = *(float2*)&in21[(xi-1)*(int)nInputPlanes + lid2];
					} else {
						*(float2*)&in_block0[bi*(int)nInputPlanes + lid2] = *(float2*)&in01[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block1[bi*(int)nInputPlanes + lid2] = *(float2*)&in11[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block2[bi*(int)nInputPlanes + lid2] = *(float2*)&in21[xi*(int)nInputPlanes + lid2];
					}
				}

				{
					int xi = xi0-1;
					if (xi == -1) {
						*(float2*)&in_block0[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in01[lid2];
						*(float2*)&in_block1[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in11[lid2];
						*(float2*)&in_block2[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in21[lid2];
					} else {
						*(float2*)&in_block0[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in01[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block1[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in11[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block2[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in21[xi*(int)nInputPlanes + lid2];
					}
				}
			}
			__syncthreads();

			if (0 && rem >= BLOCK_SIZE) {
#if 0

#define DECL_PTR(y,x)		float *p##y##x = &in_block##y[nInputPlanes * (x-1)];

				UNROLL10x3(DECL_PTR);

				float sum00 = 0;
				float sum01 = 0;
				float sum02 = 0;
				float sum03 = 0;
				float sum04 = 0;
				float sum05 = 0;
				float sum06 = 0;
				float sum07 = 0;

				float sum10 = 0;
				float sum11 = 0;
				float sum12 = 0;
				float sum13 = 0;
				float sum14 = 0;
				float sum15 = 0;
				float sum16 = 0;
				float sum17 = 0;

				{
					const float *w0 = weight + lid;

					for (int ip = 0; ip < nInputPlanes; ip++) {
#define LOAD_INPUT2(y,x)			float2 i##y##x##_2 = *(float2*)&p##y##x[ip];

						UNROLL10x3(LOAD_INPUT2);

#define LOAD_COEF(X)							\
						float w0_##X = w[X * 128]; \
						float w1_##X = w[X * 128];

#define CALC(IDX,Y,I0,I1,I2,I3,I4,I5,I6,I7)				\
						sum0 += w_##IDX * i##Y##I0; \
						sum1 += w_##IDX * i##Y##I1; \
						sum2 += w_##IDX * i##Y##I2; \
						sum3 += w_##IDX * i##Y##I3; \
						sum4 += w_##IDX * i##Y##I4; \
						sum5 += w_##IDX * i##Y##I5; \
						sum6 += w_##IDX * i##Y##I6; \
						sum7 += w_##IDX * i##Y##I7;


						{
#define LOAD_INPUT1X(Y,X)				float i##Y##X = i##Y##X##_2.x;

							UNROLL10x3(LOAD_INPUT1X);

							const float *w = (w0 + (ip * 128) * 9);
							UNROLL9(LOAD_COEF);

							{
								CALC(0,0,0,1,2,3,4,5,6,7);
								CALC(1,0,1,2,3,4,5,6,7,8);
								CALC(2,0,2,3,4,5,6,7,8,9);

								CALC(3,1,0,1,2,3,4,5,6,7);
								CALC(4,1,1,2,3,4,5,6,7,8);
								CALC(5,1,2,3,4,5,6,7,8,9);

								CALC(6,2,0,1,2,3,4,5,6,7);
								CALC(7,2,1,2,3,4,5,6,7,8);
								CALC(8,2,2,3,4,5,6,7,8,9);
							}
						}

						ip++;
						{
#define LOAD_INPUT1Y(Y,X)				float i##Y##X = i##Y##X##_2.y;

							UNROLL10x3(LOAD_INPUT1Y);

							const float *w = (w0 + (ip * 128) * 9);
							UNROLL9(LOAD_COEF);

							{
								CALC(0,0,0,1,2,3,4,5,6,7);
								CALC(1,0,1,2,3,4,5,6,7,8);
								CALC(2,0,2,3,4,5,6,7,8,9);

								CALC(3,1,0,1,2,3,4,5,6,7);
								CALC(4,1,1,2,3,4,5,6,7,8);
								CALC(5,1,2,3,4,5,6,7,8,9);

								CALC(6,2,0,1,2,3,4,5,6,7);
								CALC(7,2,1,2,3,4,5,6,7,8);
								CALC(8,2,2,3,4,5,6,7,8,9);
							}
						}

					}

#define RELU(BI)							\
					{				\
						float *out = packed_output + (yi*wsz + (xi0+BI))*nOutputPlanes; \
									\
						{			\
							int opIndex = lid; \
							float v = sum##BI; \
							v += bv;	\
									\
							float mtz = max(v, 0.0f); \
							float ltz = min(v, 0.0f); \
									\
							v = ltz * 0.1f + mtz; \
									\
							out[opIndex] = v; \
						}			\
					}

					UNROLL8(RELU);
				}
#endif
			} else {
				for (int bi=0; bi<BLOCK_SIZE; bi++) {
					int xi = xi0+bi;
					if (xi == wsz) {
						break;
					}

					const float *w0 = weight + lid*2;

					float sum0 = 0;
					float sum1 = 0;

					for (int ip=0; ip<nInputPlanes; ip++) {
						float i00, i01, i02;
						float i10, i11, i12;
						float i20, i21, i22;

						i00 = in_block0[(bi-1)*nInputPlanes+ip];
						i10 = in_block1[(bi-1)*nInputPlanes+ip];
						i20 = in_block2[(bi-1)*nInputPlanes+ip];

						i01 = in_block0[bi*nInputPlanes+ip];
						i11 = in_block1[bi*nInputPlanes+ip];
						i21 = in_block2[bi*nInputPlanes+ip];

						i02 = in_block0[(bi+1)*nInputPlanes+ip];
						i12 = in_block1[(bi+1)*nInputPlanes+ip];
						i22 = in_block2[(bi+1)*nInputPlanes+ip];

						const float *w = w0;

						float2 w0 = *(float2*)&w[(9*ip+0) * 128];
						float2 w1 = *(float2*)&w[(9*ip+1) * 128];
						float2 w2 = *(float2*)&w[(9*ip+2) * 128];
						float2 w3 = *(float2*)&w[(9*ip+3) * 128];
						float2 w4 = *(float2*)&w[(9*ip+4) * 128];
						float2 w5 = *(float2*)&w[(9*ip+5) * 128];
						float2 w6 = *(float2*)&w[(9*ip+6) * 128];
						float2 w7 = *(float2*)&w[(9*ip+7) * 128];
						float2 w8 = *(float2*)&w[(9*ip+8) * 128];

						sum0 += w0.x*i00;
						sum0 += w1.x*i01;
						sum0 += w2.x*i02;

						sum0 += w3.x*i10;
						sum0 += w4.x*i11;
						sum0 += w5.x*i12;

						sum0 += w6.x*i20;
						sum0 += w7.x*i21;
						sum0 += w8.x*i22;


						sum1 += w0.y*i00;
						sum1 += w1.y*i01;
						sum1 += w2.y*i02;

						sum1 += w3.y*i10;
						sum1 += w4.y*i11;
						sum1 += w5.y*i12;

						sum1 += w6.y*i20;
						sum1 += w7.y*i21;
						sum1 += w8.y*i22;
					}

					float *out = packed_output + (yi*wsz + xi)*nOutputPlanes;

					{
						float v = sum0;
						v += bv0;

						float mtz = max(v, 0.0f);
						float ltz = min(v, 0.0f);

						v = ltz * 0.1f + mtz;
						out[op] = v;
					}

					{
						float v = sum1;
						v += bv1;

						float mtz = max(v, 0.0f);
						float ltz = min(v, 0.0f);

						v = ltz * 0.1f + mtz;
						out[op+1] = v;
					}
				}
			}
		}
	}
}

